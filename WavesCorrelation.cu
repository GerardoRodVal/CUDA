#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
extern "C" 
{
    #include <sacio.h>
    #include <sac.h>
}

#define Nsac            5
#define MAX             1024
#define GRID_DIMENSION  16
#define BLOCK_DIMENSION 8

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s line:%d \n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
  fprintf(stderr,"ERROR - No GPU card detected.\n");
  exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("            Device Number: %d\n", i);
    printf("              Device name: %s\n",            prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",            prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",            prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

__global__ void Power( hipfftComplex *Input, hipfftComplex *Output, int xlen, int npts )
{
    int ThreadPerBlock   = blockDim.x*blockDim.y*blockDim.z;
    int ThreadNumInBlock = (threadIdx.x + threadIdx.y*blockDim.x) + threadIdx.z*(blockDim.x*blockDim.y);
    int BlockNumInGrid   = (blockIdx.x + blockIdx.y*gridDim.x) + blockIdx.z*(gridDim.x*gridDim.y);

    int globalThreadNum  = ThreadNumInBlock + ThreadPerBlock*BlockNumInGrid;

    if( globalThreadNum < xlen ){
        Output[globalThreadNum].x = (Input[globalThreadNum].x * Input[globalThreadNum].x + Input[globalThreadNum].y * Input[globalThreadNum].y)/npts;
        Output[globalThreadNum].y = 0;
    }
}

__global__ void Correlation( hipfftComplex *Input, hipfftComplex *Output, int batch_id, int size, int begin )
{

    int ThreadPerBlock  = blockDim.x*blockDim.y*blockDim.z;
    int index = threadIdx.x+(blockIdx.x*ThreadPerBlock);

    Output[ index+begin ].x = Input[ index ].x*Input[ index + batch_id*size ].x  + Input[index].y*Input[ index + batch_id*size ].y;
    Output[ index+begin ].y = Input[ index ].y*Input[ index + batch_id*size ].x  - Input[index].x*Input[ index + batch_id*size ].y;
}   

__global__ void Coherence( hipfftComplex *Input, hipfftComplex *Output, int batch_id, int size, int begin )
{
    int ThreadPerBlock  = blockDim.x*blockDim.y*blockDim.z;
    int index = threadIdx.x+(blockIdx.x*ThreadPerBlock);

    Output[ index+begin ].x = powf(abs( Input[ index ].x * Input[ index+batch_id*size ].x), 2) /    Input[ index ].x * Input[ index+batch_id*size ].x;
    Output[ index+begin ].y = powf(abs( Input[ index ].y * Input[ index+batch_id*size ].y*-1), 2) / Input[ index ].y * Input[ index+batch_id*size ].y;
}

int main(int argc, char **argv) 
{
//---------------------------------------------Time event-------------------------------------------------------

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

//------------------------------------------ Kernel invocation--------------------------------------------------

    int grid_size  = GRID_DIMENSION;
    int block_size = BLOCK_DIMENSION;

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);

//--------------------------------------------settings to sac --------------------------------------------------

    int count = 0;
    int nlen, nerr, max = MAX; 
    char kname[43];
    float *data;
    float yarray[MAX];
    float beg, del;
    char Names[Nsac][43];
    
    data = (float *)malloc( Nsac*MAX*sizeof(float));

    struct dirent *de;  
    DIR *dr = opendir(".");                                                                             //open currently directory
    while ((de = readdir(dr)) != NULL)
    {
        if( strstr( de->d_name, ".sac" ) )                                                              // only sac files
        {
            strcpy( kname , de->d_name );                                                               // reading sac files
            rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;           

            if ( nerr != 0 ) 
            {   
                fprintf(stderr, "Error reading SAC file: %s\n", kname);
                exit ( nerr ) ;
            }

            printf("%s file number %i  \n", kname, count );
            strcpy( Names[count], kname );
            memcpy(&data[count*MAX], yarray, nlen*sizeof(float));
            count ++;
        }
    }

    printf("\n");
// ---------------------------------------------fft_settings---------------------------------------------------

    int DATASIZE = MAX;
    //int size_fft = DATASIZE / 2 + 1;
    int batch    = count;    
    hipfftHandle handle_forward;
    hipfftReal *Input_fft;
    hipfftComplex *Output_fft;

// -----------------------------------------------cuda_fft-----------------------------------------------------

    hipMalloc((void**)&Input_fft,  DATASIZE * batch * sizeof(hipfftReal) );
    hipMalloc((void**)&Output_fft, DATASIZE * batch * sizeof(hipfftComplex) );

    hipMemcpy(Input_fft, data, DATASIZE * batch * sizeof(float), hipMemcpyHostToDevice);
    hipfftPlan1d(&handle_forward, DATASIZE, HIPFFT_R2C, batch);
    hipfftExecR2C(handle_forward, Input_fft, Output_fft);
    
// ----------------------------------------------- Power ------------------------------------------------------

    hipfftComplex *Power_Out;
    hipMalloc((void**)&Power_Out,  DATASIZE * batch * sizeof(hipfftComplex));
    Power<<< DimGrid, DimBlock >>>(Output_fft, Power_Out, DATASIZE*batch, DATASIZE);

// --------------------------------------- Correlation and Coherence ------------------------------------------

    int BATCH = (batch*(batch-1))/2;
    int Begin = 0;

    hipfftComplex *Correlation_Out;
    hipMalloc((void**)&Correlation_Out,  DATASIZE * BATCH * sizeof(hipfftComplex));

    hipfftComplex *Coherence_Out;
    hipMalloc((void**)&Coherence_Out,    DATASIZE * BATCH * sizeof(hipfftComplex) );


    for( int floor=1; floor<batch; floor++ ){
        printf("%i \n", Begin);
        Correlation<<< batch-floor, DATASIZE >>>(Output_fft, Correlation_Out, floor, DATASIZE, Begin); 
        Coherence  <<< batch-floor, DATASIZE >>>(Output_fft, Coherence_Out,   floor, DATASIZE, Begin); 
        Begin += DATASIZE*(batch-floor);
    }

// ----------------------------------------------- cuda_fft_i ---------------------------------------------------

    hipfftHandle handle_inverse;
    hipfftReal *Output_i;
    hipfftReal *Output_i2;
    hipfftReal *Output_i3;
    hipMalloc((void**)&Output_i,  DATASIZE * batch * sizeof(hipfftReal) );
    hipMalloc((void**)&Output_i2, DATASIZE * BATCH * sizeof(hipfftReal) );
    hipMalloc((void**)&Output_i3, DATASIZE * BATCH * sizeof(hipfftReal) );

    hipfftPlan1d( &handle_inverse, DATASIZE, HIPFFT_C2R, batch);
    hipfftExecC2R(handle_inverse, Power_Out, Output_i);

    hipfftPlan1d( &handle_inverse, DATASIZE, HIPFFT_C2R, batch);
    hipfftExecC2R(handle_inverse, Correlation_Out, Output_i2);

    hipfftPlan1d( &handle_inverse, DATASIZE, HIPFFT_C2R, batch);
    hipfftExecC2R(handle_inverse, Coherence_Out, Output_i3);

// ------------------------------------------------ Print Results ----------------------------------------------------

    hipfftReal *Out_Power = (hipfftReal*)malloc( DATASIZE * batch * sizeof(hipfftReal));
    hipfftReal *Out_Corr  = (hipfftReal*)malloc( DATASIZE * BATCH * sizeof(hipfftReal)); 
    hipfftReal *Out_Coh   = (hipfftReal*)malloc( DATASIZE * BATCH * sizeof(hipfftReal)); 
 
    hipMemcpy(Out_Power, Output_i,  DATASIZE * batch * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    hipMemcpy(Out_Corr,  Output_i2, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    hipMemcpy(Out_Coh,   Output_i3, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyDeviceToHost);

    float max_corr[BATCH];
    for (int i=0; i < BATCH; i++){
        for (int j =0; j < DATASIZE; j++){
            if (Out_Corr[i*DATASIZE + j] > max_corr[i]){
                max_corr[i] = Out_Corr[i*DATASIZE + j];
            }
        }
    }

    float max_cohr[BATCH];
    for (int i=0; i < BATCH; i++){
        for (int j =0; j < DATASIZE; j++){
            if (Out_Coh[i*DATASIZE + j] > max_corr[j]){
                max_cohr[i] = Out_Coh[i*DATASIZE + j];
            }
        }
    }

    int id_v1 = 0;
    int id_v2 = 1;
    int v_id = id_v2;
    int B = 0;
    int E = batch-1;

    for( int i=0; i<batch; i++ ){
        printf("\n--------------- Power %f of file %d --------------\n", Out_Power[i*DATASIZE], i );
        for( int j=B; j<E; j++ ){
            //printf("%i  %i \n", id_v1, id_v2);
            printf("\n with file number %d Correlation = %f \n", id_v2, max_corr[j]/(2*DATASIZE*sqrt(Out_Power[id_v1*DATASIZE]*Out_Power[id_v2*DATASIZE])) );
            printf("                      Coherence = %f \n", max_cohr[j]/DATASIZE);
            id_v1 += 1;
            id_v2 += 1;
        }
        id_v1 = 0;
        id_v2 = v_id+1;
        v_id += 1;
        B += batch-(i+1);
        E += batch-(i+2);
    }

    check_gpu_card_type();

//-------------------------------------------------Finish---------------------------------------------------

    hipEventRecord( stop, 0) ;
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop);
    printf("Time: %f milliseconds\n",elapsedTime/1000);
    printf("\n");

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipfftDestroy(handle_inverse);
    hipFree(Output_i);
    hipFree(Output_i2);
    hipFree(Output_i3);
    
    hipfftDestroy(handle_forward);
    hipFree(Input_fft);
    hipFree(Output_fft);

    hipFree(Power_Out);
    hipFree(Correlation_Out);
    hipFree(Coherence_Out);

    free(data);

    hipDeviceSynchronize();
    hipDeviceReset();

    return (EXIT_SUCCESS);
}
