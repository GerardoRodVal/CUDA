#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
extern "C" 
{
	#include <sacio.h>
	#include <sac.h>
}

#define Nsac       		2
#define GRID_DIMENSION  3
#define BLOCK_DIMENSION 3
#define MAX 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s line:%d \n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
  fprintf(stderr,"ERROR - No GPU card detected.\n");
  exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("            Device Number: %d\n", i);
    printf("              Device name: %s\n",            prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",            prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",            prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

__global__ void Correlate( hipfftComplex *Input, hipfftComplex *Output, int xlen )
{
	int NumThread = blockDim.x*blockDim.y*blockDim.z;
	int idThread  = (threadIdx.x + threadIdx.y*blockDim.x) + threadIdx.z*(blockDim.x*blockDim.y);
	int BlockId   = (blockIdx.x + blockIdx.y*gridDim.x) + blockIdx.z*(gridDim.x*gridDim.y);

	int uniqueid  = idThread + NumThread*BlockId;

	/*
	if( uniqueid < xlen ){
		Output[uniqueid].x = Input[uniqueid].x;
		Output[uniqueid].y = Input[uniqueid].y;
	}*/
}

__global__ void VectorMult( hipfftComplex *Input, hipfftComplex *Output, int xlen, int npts )
{
	int NumThread = blockDim.x*blockDim.y*blockDim.z;
	int idThread  = (threadIdx.x + threadIdx.y*blockDim.x) + threadIdx.z*(blockDim.x*blockDim.y);
	int BlockId   = (blockIdx.x + blockIdx.y*gridDim.x) + blockIdx.z*(gridDim.x*gridDim.y);

	int uniqueid  = idThread + NumThread*BlockId;

	if( uniqueid < xlen ){
		Output[uniqueid].x = (Input[uniqueid].x * Input[uniqueid].x + Input[uniqueid].y * Input[uniqueid].y)/npts;
		Output[uniqueid].y = 0;
	}
}

int main(int argc, char **argv) 
{
//---------------------------------------------Time event-------------------------------------------------------
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord( start, 0 );
//------------------------------------------ Kernel invocation--------------------------------------------------
	int grid_size  = GRID_DIMENSION;
    int block_size = BLOCK_DIMENSION;

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);

//--------------------------------------------settings to sac --------------------------------------------------
  	int count = 0;
  	int nlen, nerr, max = MAX;
  	char kname[31];
	float *data;
	float yarray[MAX];
	float beg, del;

	data = (float *)malloc( Nsac*MAX*sizeof(float));

	check_gpu_card_type();

	struct dirent *de;  
	DIR *dr = opendir(".");								//open currently directory
    while ((de = readdir(dr)) != NULL)
    {
    	if( strstr( de->d_name, ".sac" ) ) 				// only sac files
		{
			strcpy( kname , de->d_name );				// reading sac files
		  	rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;

			if ( nerr != 0 ) 
			{	
			    fprintf(stderr, "Error reading SAC file: %s\n", kname);
			    exit ( nerr ) ;
			}

			memcpy(&data[count*MAX], yarray, nlen*sizeof(float));
			count ++;
		}
	}

// ---------------------------------------------fft_settings---------------------------------------------------
	int DATASIZE = MAX;
	int size_fft = DATASIZE / 2 + 1;
  	int batch    = count;    

// -----------------------------------------------cuda_fft-----------------------------------------------------
	hipfftHandle handle_forward;
  	hipfftReal *Input_fft;
  	hipfftComplex *Output_fft;
	hipMalloc((void**)&Input_fft,  DATASIZE * batch * sizeof(hipfftReal) );
	hipMalloc((void**)&Output_fft, size_fft * batch * sizeof(hipfftComplex) );

	hipMemcpy(Input_fft, data, DATASIZE * batch * sizeof(float), hipMemcpyHostToDevice);
	hipfftPlan1d(&handle_forward, DATASIZE, HIPFFT_R2C, batch);
	hipfftExecR2C(handle_forward, Input_fft, Output_fft);
	
//---------------------------------------------- Correlation --------------------------------------------------
	hipfftComplex *Corr;
	hipMalloc((void**)&Corr,  size_fft * batch * sizeof(hipfftComplex));
	VectorMult<<<DimGrid,DimBlock>>>(Output_fft, Corr, size_fft*batch, size_fft ); 																	    // vector element x element	

	hipfftComplex *Input_i;
	hipMalloc((void**)&Input_i,  size_fft * batch * sizeof(hipfftComplex));
	Correlate<<<DimGrid,DimBlock>>>(Corr, Input_i, size_fft*batch ); 				//Todos contra todos

//------------------------------------------------cuda_fft_i---------------------------------------------------
	hipfftHandle handle_inverse;
	hipfftReal *Output_i;
	hipMalloc((void**)&Output_i,  DATASIZE * batch * sizeof(hipfftReal) );

	hipfftPlan1d( &handle_inverse, DATASIZE, HIPFFT_C2R, batch);
	hipfftExecC2R(handle_inverse, Corr, Output_i);

	hipfftReal *XCorr = (hipfftReal*)malloc((DATASIZE) * batch * sizeof(hipfftReal)); 
	hipMemcpy(XCorr, Output_i, DATASIZE * batch * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for (int i=0; i<batch; i++)
		printf(" hostOutputPowerT[%d] = %f\n",i, XCorr[DATASIZE*i]/2);

	hipEventRecord( stop, 0) ;
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop);
	printf("\n Time: %f ms\n",elapsedTime);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipfftDestroy(handle_inverse);
	hipFree(Output_i);
	hipFree(Corr);

	hipfftDestroy(handle_forward);
	hipFree(Input_fft);
	hipFree(Output_fft);

	free(data);

	hipDeviceSynchronize();
	hipDeviceReset();

	return (EXIT_SUCCESS);
}


// hostOutputPowerT[0] = 771441344.000000
// hostOutputPowerT[1] = 192102768.000000


/*
	hipfftComplex *print = (hipfftComplex*)malloc( MAX*count*sizeof(hipfftComplex));
	hipMemcpy( print, Corr, MAX*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost );
	FILE *file;
	char filename[] = "Graph.dat";
	file = fopen(filename, "w");
	int l;
	for( l = 0; l<MAX*count; l++ )
		fprintf(file, "%f    %f\n", print[l].x, print[l].y);
*/
