#include "hip/hip_runtime.h"
#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
extern "C" 
{
	#include <sacio.h>
	#include <sac.h>
}

#define GRID_DIMENSION  3
#define BLOCK_DIMENSION 3

#define MAX 60001
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s line:%d \n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
  fprintf(stderr,"ERROR - No GPU card detected.\n");
  exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("            Device Number: %d\n", i);
    printf("              Device name: %s\n",            prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",            prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",            prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}


__global__ void ComplexConj( long int nelem, hipfftComplex *array )
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int bz = blockIdx.z;

	int thx = threadIdx.x;
	int thy = threadIdx.y;
	int thz = threadIdx.z;

	int NumThread = blockDim.x*blockDim.y*blockDim.z;
	int idThread  = (thx + thy*blockDim.x) + thz*(blockDim.x*blockDim.y);
	int BlockId   = (bx + by*gridDim.x) + bz*(gridDim.x*gridDim.y);

	int uniqueid  = idThread + NumThread*BlockId;

	if (uniqueid < nelem){
		array[uniqueid].y = array[uniqueid].y*-1;
 	 }
}



int NextPower2( unsigned int v )
{
	v--;
	v |= v >> 1;
	v |= v >> 2;
	v |= v >> 4;
	v |= v >> 8;
	v |= v >> 16;
	v++;
	return v;
}



float Normalize( float *array, int n1 )
{
	float Vsum;

	for (int i=0; i<n1; i++)
	    Vsum  += array[i]*array[i];
	
	return sqrt(Vsum);
}



void Files_Xcor( hipfftComplex *xcor, int batch, int size_fft )
{	
	FILE *file;
	char filename[] = "Correlation_.dat";
	int i;
	for( i = 0; i<(batch*(batch-1))/2; i++ )
	{
		filename[11] = i + '0';
		file = fopen(filename, "w");
		int j;
		for( j = 0; j < size_fft; j++ )
		{
			fprintf(file, "%f    %f\n", xcor[j].x, xcor[j].y );
		}
		fclose(file);
	}
}


void Spect( int N )
{
	FILE   *gnuplot = NULL;
	char Correlation[] = "Correlation_.dat";
	gnuplot=popen("gnuplot","w");
	fprintf(gnuplot,"set term postscript eps enhanced color\n");
	int i;
    for( i=0; i<(N*(N-1))/2; i++ )
    {
                Correlation[11] = i + '0';
                fprintf(gnuplot, "set logscale xz\n");
                fprintf(gnuplot, "set output 'graphics_fft_%i.eps'\n", i);
                fprintf(gnuplot, "plot '%s' u :2 with lines\n", Correlation);
                fprintf(gnuplot, "set output\n");
                fflush(gnuplot);
    }
	pclose(gnuplot);
}


hipfftComplex *Vector( hipfftComplex *vectorIn, int inicio, int final)
{
	hipfftComplex *Output;
	Output = (hipfftComplex*)malloc( MAX*sizeof(hipfftComplex));

	int ind = 0;
	for( int i=inicio; i<final; i++ ){
		Output[ind] = vectorIn[i];
		ind += 1;
	}	

	return Output;
}


hipfftComplex *Comparaciones( hipfftComplex *f1, hipfftComplex *f2, int N )
{
	hipfftComplex *OutputC;

	for( int i=0; i<N; i++ ){
		OutputC[i].x = f1[i].x * f2[i].x;
		OutputC[i].y = f1[i].y * f2[i].y;
	}

	return OutputC;
}	



int main(int argc, char **argv) 
{
//----------------------------------settings to sac -----------------------------------------
  	int count = 0;
  	int nlen, nerr, max = MAX;
  	char kname[31];
	float *data;
	float yarray[MAX];
	float beg, del;

	data = (float *)malloc( 20*MAX*sizeof(float));

	check_gpu_card_type();

// reading sac files
	struct dirent *de;  
	DIR *dr = opendir(".");								//open currently directory
    while ((de = readdir(dr)) != NULL)
    {
    	if( strstr( de->d_name, ".sac" ) ) 				// only sac files
		{
			strcpy( kname , de->d_name );				// reading sac files
		  	rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;

			if ( nerr != 0 ) 
			{	
			    fprintf(stderr, "Error reading SAC file: %s\n", kname);
			    exit ( nerr ) ;
			}

			memcpy(&data[count*MAX], yarray, nlen*sizeof(float));
			count ++;
		}
	}

	int n1 = nlen;
    int n2 = nlen;

    if( n1 > n2 ){
    	printf( "Reference signal S1 cannot be longer than target S2" );
        exit(0);
    }

    int nx = n2-n1+1;
	int nfft;
    nfft = NextPower2( n2+n1 );

// --------------------------------------cuda_fft---------------------------------------------------
	hipfftHandle plan;							// settings plan to fft
	hipfftComplex *data_fft;
	hipfftComplex *Out_fft;
	hipfftReal *dev_dat;

	int rank = 1;                            
  	int n[] = { nlen };                      
  	int istride = 1, ostride = 1;            
  	int idist = MAX, odist = ( nlen / 2 + 1); 
  	int inembed[] = { 0 };                   
  	int onembed[] = { 0 };                   
  	int size_fft = (nlen );
  	int batch = count;   

	gpuErrchk(hipMalloc((void**)&dev_dat, MAX*count*sizeof(hipfftReal) ));
	gpuErrchk(hipMalloc((void**)&data_fft, size_fft*count*sizeof(hipfftComplex) ));
	Out_fft = (hipfftComplex*)malloc( size_fft * count * sizeof(hipfftComplex));
	gpuErrchk(hipMemcpy(dev_dat, data, MAX*count*sizeof(float), hipMemcpyHostToDevice));
								
	hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
	hipfftExecR2C(plan, dev_dat, data_fft);

	gpuErrchk(hipMemcpy(Out_fft, data_fft, size_fft*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	

//------------------------------------Complex conjugate--------------------------------------------------------
	int grid_size  = GRID_DIMENSION;
    int block_size = BLOCK_DIMENSION;

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);

    hipfftHandle handle;

    hipfftReal *ComCon_d;
	hipfftComplex *ComCon_dO;
	hipfftComplex *Out_conj; 
	hipfftComplex *fft_conj;

	Out_conj = (hipfftComplex*)malloc((nlen) * count * sizeof(hipfftComplex));
	hipMalloc((void**)&ComCon_d, nlen*count*sizeof(hipfftReal));
	hipMalloc((void**)&ComCon_dO, (nlen) * count * sizeof(hipfftComplex));
    hipMalloc((void**)&fft_conj, (nlen) * count * sizeof(hipfftComplex));

	hipMemcpy(ComCon_d, data_fft, nlen*count*sizeof(hipfftReal), hipMemcpyDeviceToDevice);

    hipfftPlanMany(&handle, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);

	hipfftExecR2C(handle, ComCon_d, ComCon_dO);
	hipMemcpy(fft_conj, ComCon_dO, (nlen)*count*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);

	ComplexConj<<<DimGrid,DimBlock>>>( (nlen)*count, fft_conj );

	hipMemcpy(Out_conj, fft_conj, (nlen)*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

//-------------------------------------------- Correlation --------------------------------------------------------
	#define COMP (batch*(batch-1))/2

	hipfftComplex *xcor;																			// contiene todas las comparaciones 
	xcor = (hipfftComplex*)malloc( MAX*COMP*sizeof(hipfftComplex));

	int begin = 0;
	int end = MAX;
	int foot = 0;
	int n3 = 1;

	hipfftComplex *f1;
	hipfftComplex *f2;
	hipfftComplex *f1xf2;	

	f1 = (hipfftComplex*)malloc( MAX * sizeof(hipfftComplex));
	f2 = (hipfftComplex*)malloc( MAX * sizeof(hipfftComplex));
	f1xf2 = (hipfftComplex*)malloc( MAX * sizeof(hipfftComplex));


	for( int x=0; x<batch-1; x++ ){
		f1 = Vector( Out_conj, begin, end );
		int begin2 = MAX + foot;
		int end2 = (MAX+MAX) + foot;
		for( int y=0; y<batch-n3; y++ ){
			f2 = Vector( Out_fft, begin2, end2 );
			f1xf2 = Comparaciones( f1, f2, MAX );
			begin2 += MAX;
			end2 += MAX;
			memcpy( xcor, f1xf2, MAX*sizeof(hipfftComplex)  );
		}
		n3 += 1;
		begin += MAX;
		end += MAX;
		foot += MAX;
	}
//------------------------------------------------ ifft-----------------------------------------------------------

	hipfftComplex *Final_Xcor;
	Final_Xcor = (hipfftComplex*)malloc( MAX*COMP*sizeof(hipfftComplex));

	hipfftComplex *data_fft_i;
	hipMalloc((void**)&data_fft_i,  MAX*COMP*sizeof(hipfftComplex));

	hipfftHandle plan_i;							
	hipfftReal *dev_dat_i;

	int rank_i = -1;

	gpuErrchk(hipMalloc((void**)&dev_dat_i, MAX*COMP*sizeof(hipfftReal) ));
	gpuErrchk(hipMemcpy(dev_dat_i, xcor, MAX*count*sizeof(hipfftComplex), hipMemcpyHostToDevice));
									
	hipfftPlanMany(&plan_i, rank_i, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
	hipfftExecR2C(plan_i, dev_dat_i, data_fft_i);

	gpuErrchk(hipMemcpy(Final_Xcor, data_fft_i,  MAX*COMP*sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	
// ------------------------ scale by sqrt(norm(s1)*norm(s2win)) where s2win is the moving window of s2 -----------

	float s1[n1];
	float s2[n2];
	float s2s2[n2];
	float scal[nx] = {0};											//     scal = np.zeros(nx)

	int size = MAX;
	for( int i=0; i<batch-1; i++ ){

	   	int ind=0;
		for( int j=(size-MAX); j<size; j++ ){				
			s1[ind] = data[j+MAX]; 
			s2[ind] = data[j];
			s2s2[ind] = s2[ind]*s2[ind];
			ind += 1;
		}
		size = size+MAX;

		for( int k=0; k<n1; k++  ){
			scal[0] += s2s2[k];										//    scal[0] = sum(s2s2[:n1])
		}

		for( int l=0; l<nx-1; l++ ){							
			scal[l+1] = scal[l] + s2s2[n1+l]-s2s2[l];				
		}

		float norm;
		norm = Normalize(s1, n1);
			
		for( int m=0; m<nx; m++ ){                              	//     xcor = xcor[:nx]
			scal[m] = sqrt(scal[m]) * norm;
			Final_Xcor[m].x = Final_Xcor[m].x/scal[m];
		}

		Files_Xcor( Final_Xcor, batch, size_fft );
		Spect( batch );
	}

	hipfftDestroy(handle);
	hipFree(Out_fft);
	hipFree(Out_conj);
	hipFree(ComCon_dO);
	hipFree(ComCon_d);
	hipFree(fft_conj);
	hipFree(dev_dat);
	hipFree(dev_dat_i);
	hipFree(data_fft);
	hipFree(data_fft_i);
	hipfftDestroy(plan);
	hipfftDestroy(plan_i);
	free(data);
	hipDeviceSynchronize();
	hipDeviceReset();
	return (EXIT_SUCCESS);

}