#include "hip/hip_runtime.h"
#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
extern "C" 
{
	#include <sacio.h>
	#include <sac.h>
}

//#define DATASIZE 8
//#define BATCH 3
#define GRID_DIMENSION  3
#define BLOCK_DIMENSION 3

#define MAX 60001
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


__global__ void GPUmemo( float *data, int pts )
{
	__shared__ float* trace;

	trace = (float *)malloc(pts*sizeof(float));
	int Blocks;
	for( Blocks = 0; Blocks < gridDim.x; Blocks++ )
	{
		trace[threadIdx.x] = data[threadIdx.x + Blocks*pts];
	}
}


__global__ void ComplexConj( long int nelem, hipfftComplex *array )
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int bz = blockIdx.z;

	int thx = threadIdx.x;
	int thy = threadIdx.y;
	int thz = threadIdx.z;

	int NumThread = blockDim.x*blockDim.y*blockDim.z;
	int idThread  = (thx + thy*blockDim.x) + thz*(blockDim.x*blockDim.y);
	int BlockId   = (bx + by*gridDim.x) + bz*(gridDim.x*gridDim.y);

	int uniqueid  = idThread + NumThread*BlockId;

	if (uniqueid < nelem){
		array[uniqueid].y = array[uniqueid].y*-1;
 	 }
}


void ComplexGraph( hipfftComplex *data, int size_fft)
{
	FILE *file;
	char filename[] = "ComplexData.dat";
	file = fopen(filename, "w");
	int l;
	for( l = 0; l<size_fft; l++ )
		fprintf(file, "%f    %f\n", data[l].x, data[l].y);

	FILE *gnuplot = NULL;
	gnuplot=popen("gnuplot","w");
	fprintf(gnuplot,"set term postscript eps enhanced color\n");
	fprintf(gnuplot, "set logscale xz\n");
	fprintf(gnuplot, "set output 'graphic_Complex.eps'\n");
	fprintf(gnuplot, "plot '%s' u 2 with lines\n", filename);
	fprintf(gnuplot, "set output\n");
	fflush(gnuplot);
	pclose(gnuplot);
	fclose(file);

}



inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s line:%d \n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
  fprintf(stderr,"ERROR - No GPU card detected.\n");
  exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("            Device Number: %d\n", i);
    printf("              Device name: %s\n",            prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",            prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",            prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}



void Fourier( hipfftComplex *fft, int batch, int size_fft )
{	
	FILE *file;
	char filename[] = "Array_.dat";
	int i;
	for( i = 0; i<batch; i++ )
	{
		filename[5] = i + '0';
		file = fopen(filename, "w");
		int j;
		for( j = 0; j < size_fft; j++ )
		{
			float result1 = fft[i*size_fft + j].x;
			float result2 = fft[i*size_fft + j].y;
			float result3 = sqrt(fft[i*size_fft + j].x*fft[i*size_fft + j].x + fft[i*size_fft + j].y*fft[i*size_fft + j].y);
			fprintf(file, "%f    %f    %f\n", result1, result2, result3 );
		}
		fclose(file);
	}
}


void Spect( int N )
{
	FILE   *gnuplot = NULL;
	char fft_file[] = "Array_.dat";
	gnuplot=popen("gnuplot","w");
	fprintf(gnuplot,"set term postscript eps enhanced color\n");
	int i;
    for( i=0; i<N; i++ )
    {
                fft_file[5] = i + '0';
                fprintf(gnuplot, "set logscale xz\n");
                fprintf(gnuplot, "set output 'graphics_fft_%i.eps'\n", i);
                fprintf(gnuplot, "plot '%s' u :(log($3)) with lines\n", fft_file);
                fprintf(gnuplot, "set output\n");
                fflush(gnuplot);

                fft_file[5] = i + '0';
                fprintf(gnuplot, "set logscale xz\n");
                fprintf(gnuplot, "set output 'graphics_%i.eps'\n", i);
                fprintf(gnuplot, "plot '%s' u :3 with lines\n", fft_file);
                fprintf(gnuplot, "set output\n");
                fflush(gnuplot);
    }
	pclose(gnuplot);
}


int main(int argc, char **argv) 
{
//----------------------------------settings to sac -----------------------------------------
  	int count = 0;
  	int nlen, nerr, max = MAX;
  	char kname[31];
	float *data;
	float yarray[MAX];
	float beg, del;

	data = (float *)malloc( 20*MAX*sizeof(float));	

	check_gpu_card_type();

// reading sac files
	struct dirent *de;  
	DIR *dr = opendir(".");								//open currently directory
    while ((de = readdir(dr)) != NULL)
    {
    	if( strstr( de->d_name, ".sac" ) ) 				// only sac files
		{
			strcpy( kname , de->d_name );				// reading sac files
		  	rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;

			if ( nerr != 0 ) 
			{	
			    fprintf(stderr, "Error reading SAC file: %s\n", kname);
			    exit ( nerr ) ;
			}

			memcpy(&data[count*MAX], yarray, nlen*sizeof(float));
			count ++;
		}
	}

// --------------------------------------cuda_fft---------------------------------------------------
	hipfftHandle plan;							// settings plan to fft
	hipfftComplex *data_fft;
	hipfftComplex *outfft;
	hipfftReal *dev_dat;

	int rank = 1;                            
  	int n[] = { nlen };                      
  	int istride = 1, ostride = 1;            
  	int idist = MAX, odist = (nlen / 2 + 1); 
  	int inembed[] = { 0 };                   
  	int onembed[] = { 0 };                   
  	int size_fft = (nlen / 2 + 1);
  	int batch = count;   

	gpuErrchk(hipMalloc((void**)&dev_dat, MAX*count*sizeof(hipfftReal) ));
	gpuErrchk(hipMalloc((void**)&data_fft, size_fft*count*sizeof(hipfftComplex) ));
	outfft = (hipfftComplex*)malloc( size_fft * count * sizeof(hipfftComplex));
	gpuErrchk(hipMemcpy(dev_dat, data, MAX*count*sizeof(float), hipMemcpyHostToDevice));
								
	hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
	hipfftExecR2C(plan, dev_dat, data_fft);

//------------------------------------Complex conjugate--------------------------------------------------------
	int grid_size  = GRID_DIMENSION;
    int block_size = BLOCK_DIMENSION;

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);

    hipfftHandle handle;

    hipfftReal *ComCon_d;
	//hipfftReal *ComCon_h;
	hipfftComplex *ComCon_dO;
	hipfftComplex *ComCon_hO; 
	hipfftComplex *fft_conj;

	//ComCon_h = (hipfftReal*)malloc(nlen*count*sizeof(hipfftReal));
	ComCon_hO = (hipfftComplex*)malloc((nlen) * count * sizeof(hipfftComplex));
	hipMalloc((void**)&ComCon_d, nlen*count*sizeof(hipfftReal));
	hipMalloc((void**)&ComCon_dO, (nlen) * count * sizeof(hipfftComplex));
    hipMalloc((void**)&fft_conj, (nlen) * count * sizeof(hipfftComplex));

/*    for (int i=0; i<BATCH; i++)
        for (int j=0; j<DATASIZE; j++){ 
		ComCon_h[i*DATASIZE + j] = (hipfftReal)((i + 1) + j);
	//	printf("ComCon_h[%d]=%f\n",i*DATASIZE + j, ComCon_h[i*DATASIZE + j]);
	}
*/
	hipMemcpy(ComCon_d, data_fft, nlen*count*sizeof(hipfftReal), hipMemcpyDeviceToDevice);

    hipfftPlanMany(&handle, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);

	hipfftExecR2C(handle, ComCon_d, ComCon_dO);
	hipMemcpy(fft_conj, ComCon_dO, (nlen)*count*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);

	ComplexConj<<<DimGrid,DimBlock>>>( (nlen)*count, fft_conj );

	hipMemcpy(ComCon_hO, fft_conj, (nlen)*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

/*
	for (int i=0; i<BATCH; i++)
        for (int j=0; j<(DATASIZE / 2 + 1); j++)
            printf("Batch  = %i j= %i real %f imag %f\n", i, j, ComCon_hO[i*(DATASIZE / 2 + 1) + j].x, ComCon_hO[i*(DATASIZE / 2 + 1) + j].y);
*/	
	ComplexGraph( ComCon_hO, size_fft );

	hipfftDestroy(handle);
    gpuErrchk(hipFree(ComCon_dO));
    gpuErrchk(hipFree(ComCon_d));
    gpuErrchk(hipFree(fft_conj));

//-------------------------------------------------------------------------------------------------------------

    gpuErrchk(hipMemcpy(outfft, data_fft, size_fft*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	Fourier( outfft, batch, size_fft );
	Spect( batch );

	GPUmemo<<<count,nlen>>>( dev_dat, nlen );


	hipFree(dev_dat);
	hipFree(data_fft);
	hipfftDestroy(plan);
	free(data);
	hipDeviceSynchronize();
	hipDeviceReset();
  	return (EXIT_SUCCESS);
}