#include "hip/hip_runtime.h"
#include <stdio.h>
#include <dirent.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
extern "C" 
{
	#include <sacio.h>
	#include <sac.h>
}

#define MAX 60001
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s line:%d \n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
  fprintf(stderr,"ERROR - No GPU card detected.\n");
  exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("            Device Number: %d\n", i);
    printf("              Device name: %s\n",            prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",            prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",            prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}




__global__ void GPUmemo( float *data, int pts )
{
	__shared__ float* trace;

	trace = (float *)malloc(pts*sizeof(float));
	int Blocks;
	for( Blocks = 0; Blocks < gridDim.x; Blocks++ )
	{
		trace[threadIdx.x] = data[threadIdx.x + Blocks*pts];
	}
}



void Fourier( hipfftComplex *fft, int batch, int size_fft )
{	
	FILE *file;
	char filename[] = "Array_.dat";
	int i;
	for( i = 0; i<batch; i++ )
	{
		filename[5] = i + '0';
		file = fopen(filename, "w");
		int j;
		for( j = 0; j < size_fft; j++ )
		{
			float result1 = fft[i*size_fft + j].x;
			float result2 = fft[i*size_fft + j].y;
			float result3 = sqrt(fft[i*size_fft + j].x*fft[i*size_fft + j].x + fft[i*size_fft + j].y*fft[i*size_fft + j].y);
			fprintf(file, "%f    %f    %f\n", result1, result2, result3 );
		}
		fclose(file);
	}
}


void Spect( int N )
{
	FILE   *gnuplot = NULL;
	//char fft_file[] = "output_.dat";
	char fft_file[] = "Array_.dat";
	gnuplot=popen("gnuplot","w");
	fprintf(gnuplot,"set term postscript eps enhanced color\n");
	int i;
    for( i=0; i<N; i++ )
    {
                fft_file[5] = i + '0';
                fprintf(gnuplot, "set logscale xz\n");
                fprintf(gnuplot, "set output 'graphics_fft_%i.eps'\n", i);
                fprintf(gnuplot, "plot '%s' u :(log($3)) with lines\n", fft_file);
                fprintf(gnuplot, "set output\n");
                fflush(gnuplot);
    }
	pclose(gnuplot);
}


int main(int argc, char **argv) 
{
//----------------------------------settings to sac -----------------------------------------
  	int count = 0;
  	int nlen, nerr, max = MAX;
  	char kname[31];
	float *data;
	float yarray[MAX];
	float beg, del;

	data = (float *)malloc( 20*MAX*sizeof(float));	

	check_gpu_card_type();
// reading sac files
	struct dirent *de;  
	DIR *dr = opendir(".");								//open currently directory
    while ((de = readdir(dr)) != NULL)
    {
    	if( strstr( de->d_name, ".sac" ) ) 				// only sac files
		{
			strcpy( kname , de->d_name );				// reading sac files
		  	rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;

			if ( nerr != 0 ) 
			{	
			    fprintf(stderr, "Error reading SAC file: %s\n", kname);
			    exit ( nerr ) ;
			}

			memcpy(&data[count*MAX], yarray, nlen*sizeof(float));
			count ++;
		}
	}

	
//-------------------------------------------------------------------------------------------------	
/*	FILE *fileMat;
	fileMat = fopen("data.dat","w");
	for (int j = 0; j < 20*MAX; j++)
		fprintf(fileMat,"%f\n",data[j]);
*/

// --------------------------------------cuda_fft---------------------------------------------------
	hipfftHandle plan;							// settings plan to fft
	hipfftComplex *data_fft;
	hipfftComplex *outfft;
	hipfftReal *dev_dat;

	int rank = 1;                            
  	int n[] = { nlen };                      
  	int istride = 1, ostride = 1;            
  	int idist = MAX, odist = (nlen / 2 + 1); 
  	int inembed[] = { 0 };                   
  	int onembed[] = { 0 };                   
  	int size_fft = (nlen / 2 + 1);
  	int batch = count;   

	gpuErrchk(hipMalloc((void**)&dev_dat, MAX*count*sizeof(hipfftReal) ));
	gpuErrchk(hipMalloc((void**)&data_fft, size_fft*count*sizeof(hipfftComplex) ));
	outfft = (hipfftComplex*)malloc( size_fft * count * sizeof(hipfftComplex));
	gpuErrchk(hipMemcpy(dev_dat, data, MAX*count*sizeof(float), hipMemcpyHostToDevice));
								
	hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, batch);
	hipfftExecR2C(plan, dev_dat, data_fft);

	gpuErrchk(hipMemcpy(outfft, data_fft, size_fft*count*sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	

	printf(" ********** CONFG *********\n");
  	printf(" rank     = %d\n", rank       );
  	printf(" n[0]     = %d\n", n[0]       );
  	printf(" inembed  = %d\n", inembed[0] );
  	printf(" istride  = %d\n", istride    );
  	printf(" onembed  = %d\n", onembed[0] );
  	printf(" ostride  = %d\n", ostride    );
  	printf(" odist    = %d\n", odist      );
  	printf(" batch    = %d\n", batch      );
  	printf(" count    = %d\n", count      );
	printf(" size_fft = %d\n", size_fft   );
  	printf(" **************************\n");

	//printf(" %i %i\n",  outfft[0].x, outfft[0].y );
	Fourier(outfft, batch, size_fft);
	Spect( batch );

	GPUmemo<<<count,nlen>>>( dev_dat, nlen );

	hipFree(dev_dat);
	hipFree(data_fft);
	hipfftDestroy(plan);
	free(data);
	hipDeviceReset();
  	return (EXIT_SUCCESS);
}
